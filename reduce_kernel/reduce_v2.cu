#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


//v3: 利用空闲线程 (0.164864 ms)
template<int blockSize>
__global__ void reduce_v3(float * d_in, float * d_out) {
    __shared__ float smem[blockSize];

    uint32_t tid = threadIdx.x;

    // cross block
    uint32_t gtid = blockIdx.x * (2 * blockDim.x) + threadIdx.x;
    smem[tid] = d_in[gtid] + d_in[gtid + blockSize];

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}
bool checkResult(const float* out, const float groundtruth, int n) {
    float sum = 0.0f;
    for (int i = 0; i < n; ++i) {
        sum += out[i];
    }
    if (sum != groundtruth) return false;
    return true;
}

int main() {
    float milliseconds = 0;
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 256;
    int gridSize = std::min((N + blockSize - 1) / blockSize, deviceProp.maxGridSize[0]);
    // maxGridSize = 100000

    float* a = (float *)malloc(N * sizeof(float));
    float* d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float* out = (float *)malloc(gridSize * sizeof(float));
    float* d_out;
    hipMalloc((void **)&d_out, gridSize * sizeof(float));

    for (int i = 0; i < N; ++i) {
        a[i] = 1.0f;
    }

    float groundtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(gridSize);
    dim3 Block(blockSize / 2);

    // warmup 
    reduce_v3<blockSize / 2><<<Grid, Block>>>(d_a, d_out);
    hipDeviceSynchronize();
    
    // kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v3<blockSize / 2><<<Grid, Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(out, d_out, gridSize * sizeof(float), hipMemcpyDeviceToHost);

    printf("allocated %d blocks, data counts are %d \n", gridSize, N);
    bool is_right = checkResult(out, groundtruth, gridSize);
    if (is_right) {
        printf("the answer is right \n");
    }
    else {
        printf("the answer is wrong \n");
        // for (int i = 0; i < gridSize; i++) {
        //     printf("res per block : %f ", out[i]);
        // }
    }
    printf("reduce baseline latency is %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}