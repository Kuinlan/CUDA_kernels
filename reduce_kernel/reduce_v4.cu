#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


//v4: 线程束级展开
//v5: 线程块级完全展开 (0.165728 ms)
template<int blockSize>
__device__ void blockReduce(float * sdata) {
    if (blockSize >= 1024) {
        if (threadIdx.x < 512) {
            sdata[threadIdx.x] += sdata[threadIdx.x + 512];
        }
    }
    __syncthreads();
    if (blockSize >= 512) {
        if (threadIdx.x < 256) {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
    }
    __syncthreads();
    if (blockSize >= 256) {
        if (threadIdx.x < 128) {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
    }
    __syncthreads();
    if (blockSize >= 128) {
        if (threadIdx.x < 64) {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
    }
    __syncthreads();
}

__device__ void warpReduce(volatile float* smem, int tid) {
    float x = smem[tid];
    if (blockDim.x >= 64) {
        // 分离读写操作，先读后写
        x += smem[tid + 32]; __syncwarp();
        smem[tid] = x; __syncwarp();
    }
    x += smem[tid + 16]; __syncwarp();
    smem[tid] = x; __syncwarp();

    x += smem[tid + 8]; __syncwarp();
    smem[tid] = x; __syncwarp();

    x += smem[tid + 4]; __syncwarp();
    smem[tid] = x; __syncwarp();

    x += smem[tid + 2]; __syncwarp();
    smem[tid] = x; __syncwarp();

    x += smem[tid + 1]; __syncwarp();
    smem[tid] = x; __syncwarp();
}

template<int blockSize>
__global__ void reduce_v4(float * d_in, float * d_out) {
    __shared__ float smem[blockSize];

    uint32_t tid = threadIdx.x;

    // cross block
    uint32_t gtid = blockIdx.x * (2 * blockDim.x) + threadIdx.x;
    smem[tid] = d_in[gtid] + d_in[gtid + blockSize];

    // for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
    //     if (tid < stride) {
    //         smem[tid] += smem[tid + stride];
    //     }
    //     __syncthreads();
    // }
    blockReduce<blockSize>(smem);

    if (tid < 32) {
        warpReduce(smem, tid);
    }
    
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}
bool checkResult(const float* out, const float groundtruth, int n) {
    float sum = 0.0f;
    for (int i = 0; i < n; ++i) {
        sum += out[i];
    }
    if (sum != groundtruth) return false;
    return true;
}

int main() {
    float milliseconds = 0;
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 256;
    int gridSize = std::min((N + blockSize - 1) / blockSize, deviceProp.maxGridSize[0]);
    // maxGridSize = 100000

    float* a = (float *)malloc(N * sizeof(float));
    float* d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float* out = (float *)malloc(gridSize * sizeof(float));
    float* d_out;
    hipMalloc((void **)&d_out, gridSize * sizeof(float));

    for (int i = 0; i < N; ++i) {
        a[i] = 1.0f;
    }

    float groundtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(gridSize);
    dim3 Block(blockSize / 2);

    // warmup 
    reduce_v4<blockSize / 2><<<Grid, Block>>>(d_a, d_out);
    hipDeviceSynchronize();
    
    // kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v4<blockSize / 2><<<Grid, Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(out, d_out, gridSize * sizeof(float), hipMemcpyDeviceToHost);

    printf("allocated %d blocks, data counts are %d \n", gridSize, N);
    bool is_right = checkResult(out, groundtruth, gridSize);
    if (is_right) {
        printf("the answer is right \n");
    }
    else {
        printf("the answer is wrong \n");
        // for (int i = 0; i < gridSize; i++) {
        //     printf("res per block : %f ", out[i]);
        // }
    }
    printf("reduce baseline latency is %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}