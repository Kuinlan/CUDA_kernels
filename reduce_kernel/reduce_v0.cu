#include <hip/hip_runtime.h>

#include <bits/stdc++.h>

// v0: shared memory  
// latency: 0.441280 ms (相邻), 0.266016 ms （交错） 

template<int blockSize>
__global__ void reduce_v0(const float* d_in, float* d_out, int size) {
    __shared__ float smem[blockSize];
    int tid = threadIdx.x;
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    smem[tid] = d_in[gtid];
    __syncthreads();

    // neiborhood1: 0.303 ms 
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        // 没有 thread divergence，因为没有影响达到 reconvergence 的时间
        if ((tid & (stride * 2 - 1)) == 0) {  // 0.304 ms
        // if (tid % (stride * 2) == 0) { // 0.44 ms
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    // // neiborhood2: 0.299ms
    // for (int stride = 1; stride < blockDim.x; stride *= 2) {
    //     int index = 2 * stride * tid;
    //     if (index < blockDim.x) {  // 0.304 ms
    //         smem[index] += smem[index + stride];
    //     }
    //     __syncthreads();
    // }

    // // interleave
    // for (int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
    //     if (tid < stride) {
    //         smem[tid] += smem[tid + stride];
    //     }
    //     __syncthreads();
    // }

    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool checkResult(const float* out, const float groundtruth, int n) {
    float sum = 0.0f;
    for (int i = 0; i < n; ++i) {
        sum += out[i];
    }
    if (sum != groundtruth) return false;
    return true;
}


int main() {
    float milliseconds = 0;
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 256;
    int gridSize = std::min((N + blockSize - 1) / blockSize, deviceProp.maxGridSize[0]);
    // maxGridSize = 100000

    float* a = (float *)malloc(N * sizeof(float));
    float* d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float* out = (float *)malloc(gridSize * sizeof(float));
    float* d_out;
    hipMalloc((void **)&d_out, gridSize * sizeof(float));

    for (int i = 0; i < N; ++i) {
        a[i] = 1.0f;
    }

    float groundtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(gridSize);
    dim3 Block(blockSize);

    // warmup 
    reduce_v0<blockSize><<<Grid, Block>>>(d_a, d_out, N);
    hipDeviceSynchronize();
    
    // kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v0<blockSize><<<Grid, Block>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(out, d_out, gridSize * sizeof(float), hipMemcpyDeviceToHost);

    printf("allocated %d blocks, data counts are %d \n", gridSize, N);
    bool is_right = checkResult(out, groundtruth, gridSize);
    if (is_right) {
        printf("the answer is right \n");
    }
    else {
        printf("the answer is wrong \n");
        // for (int i = 0; i < gridSize; i++) {
        //     printf("res per block : %f ", out[i]);
        // }
    }
    printf("reduce baseline latency is %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}