#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// baseline 版本
// latency: 452ms 
// single device thread processing
__global__ void reduce_baseline(const int* input, int* output, uint32_t size) {
    int sum = 0;
    for (int i = 0; i < size; ++i) {
        sum += input[i];
    }
    *output = sum;
}

__global__ void reduce_baseline_warmup(const int* input, int* output, uint32_t size) {
    int sum = 0;
    for (int i = 0; i < size; ++i) {
        sum += input[i];
    }
    *output = sum;
}

bool CheckResult(int *out, int groundtruth, int n) {
    for (int i = 0; i < n; i++) {
        if (out[i] != groundtruth) return false;
    }
    return true;
}

int main() {
    float milliseconds = 0;
    const int N = 25600000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 1;
    int gridSize = 1;

    int* a = (int *)malloc(N * sizeof(int));
    int* d_a;
    hipMalloc((void **)&d_a, N * sizeof(int));

    int* out = (int *)malloc(gridSize * sizeof(int));
    int* d_out;
    hipMalloc((void **)&d_out, gridSize * sizeof(int));

    for (int i = 0; i < N; ++i) {
        a[i] = 1;
    }

    int groundtruth = N * 1;

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 Grid(gridSize);
    dim3 Block(blockSize);

    // warmup 
    reduce_baseline_warmup<<<Grid, Block>>>(d_a, d_out, N);
    hipDeviceSynchronize();
    
    // kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_baseline<<<Grid, Block>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(out, d_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);
    printf("allocated %d blocks, data counts are %d \n", gridSize, N);
    bool is_right = CheckResult(out, groundtruth, gridSize);
    if (is_right) {
        printf("the answer is right \n");
    }
    else {
        printf("the answer is wrong \n");
        for (int i = 0; i < gridSize; i++) {
            printf("res per block : %lf ", out[i]);
        }
    }
    printf("reduce baseline latency is %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}