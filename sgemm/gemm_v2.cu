// sgemm one thread for muti data point without prefetch
// blocks of A is stored in smem transposed

#include <stdio.h>
#include <stdlib.h>
#include "assert.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

// K: ldA
// N: ldB
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X  // width of block of C that each thread calculate
>
__global__ void Sgemm(
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C,
    const int M,
    const int N,
    const int K) {

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int tid = ty * blockDim.x + tx;

  // smem
  __shared__ float smemA[BLOCK_SIZE_K][BLOCK_SIZE_M];  // transposed A
  __shared__ float smemB[BLOCK_SIZE_K][BLOCK_SIZE_N];

  // register for C
  float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};

  //register for storing fragment of A and B
  float frag_a [THREAD_SIZE_Y];
  float frag_b [THREAD_SIZE_X];

  // one thread writes mutiple data points
  const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4; // a thread load 4 data from HBM to smem
  const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4; // 32

  // row number and col number that needs to be loaded by this thread
  const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
  const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

  const int A_TILE_COL = (tid % A_TILE_THREAD_PER_ROW) * 4;
  const int B_TILE_COL = (tid % B_TILE_THREAD_PER_ROW) * 4;

  const int A_TILE_ROW_STRIDE = (blockDim.x * blockDim.y) / A_TILE_THREAD_PER_ROW; // 8
  const int B_TILE_ROW_STRIDE = (blockDim.x * blockDim.y) / B_TILE_THREAD_PER_ROW; // 8

  // local address for a block to perform k-split
  A = &A[BLOCK_SIZE_M * by * K];
  B = &B[BLOCK_SIZE_N * bx];

  #pragma unroll
  // K split
  for (int k = 0; k < K; k += BLOCK_SIZE_K) {
    // load from global to smem
    // block size stride
    // 针对线程块中的线程需要多批次地将数据从 gmem 搬运到 smem, 循环次数等于搬运次数
    #pragma unroll
    for (int i = A_TILE_ROW_START; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
      // note A is transposed after loading
      // use register to buffer
      float4 buffer = FETCH_FLOAT4(A[OFFSET(i, A_TILE_COL + k, K)]);
      smemA[A_TILE_COL][i] = buffer.x;
      smemA[A_TILE_COL + 1][i] = buffer.y;
      smemA[A_TILE_COL + 2][i] = buffer.z;
      smemA[A_TILE_COL + 3][i] = buffer.w;
    }

    #pragma unroll
    for (int i = B_TILE_ROW_START; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
      FETCH_FLOAT4(smemB[i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(i + k, B_TILE_COL, N)]);
    }

    __syncthreads(); // 保证读写顺序

    // 一个线程计算 [THREAD_SIZE_Y, THREAD_SIZE_X] 个数据
    #pragma unroll
    for (int i = 0; i < BLOCK_SIZE_K; ++i) {
      #pragma unroll
      for (int j = 0; j < THREAD_SIZE_Y; j += 4) {
        FETCH_FLOAT4(frag_a[j]) = FETCH_FLOAT4(smemA[i][ty * THREAD_SIZE_Y + j]);
      }
      #pragma unroll
      for (int j = 0; j < THREAD_SIZE_X; j += 4) {
        FETCH_FLOAT4(frag_b[j]) = FETCH_FLOAT4(smemB[i][tx * THREAD_SIZE_X + j]);
      }
      #pragma unroll
      for (int y = 0; y < THREAD_SIZE_Y; ++y) {
        #pragma unroll
        for (int x = 0; x < THREAD_SIZE_X; ++x) {
          accum[y][x] += frag_a[y] * frag_b[x];
        }
      }
    }
    __syncthreads();
  }
  // write from register back to C
#pragma unroll
  for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
#pragma unroll
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
      FETCH_FLOAT4(C[OFFSET(by * BLOCK_SIZE_M + ty * THREAD_SIZE_Y + thread_y,
                            bx * BLOCK_SIZE_N + tx * THREAD_SIZE_X + thread_x,
                            N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
    }
  }

}

int main(int argc, char** argv) {
  if (argc != 4) {
    printf("usage: ./main [M] [K] [N]\n");
    exit(0);
  }
  size_t M = atoi(argv[1]);
  size_t K = atoi(argv[2]);
  size_t N = atoi(argv[3]);

  assert( M%8 == 0);
  assert( N%8 == 0);
  assert( K%8 == 0);

  size_t bytes_A = sizeof(float) * M * K;
  size_t bytes_B = sizeof(float) * K * N;
  size_t bytes_C = sizeof(float) * M * N;
  float* h_A = (float*)malloc(bytes_A);
  float* h_B = (float*)malloc(bytes_B);
  float* h_C = (float*)malloc(bytes_C);
  float* h_C1 = (float*)malloc(bytes_C);

  float* d_A;
  float* d_B;
  float* d_C;

  checkCudaErrors(hipMalloc(&d_A, bytes_A));
  checkCudaErrors(hipMalloc(&d_B, bytes_B));
  checkCudaErrors(hipMalloc(&d_C, bytes_C));
  double msecPerMatrixMul[2] = {0, 0};
  double gigaFlops[2] = {0, 0};
  double flopsPerMatrixMul = 2.0 * M * N * K;

  const int BLOCK_SIZE_M = 128;
  const int BLOCK_SIZE_K = 8;
  const int BLOCK_SIZE_N = 128;
  const int THREAD_SIZE_X = 8;
  const int THREAD_SIZE_Y = 8;

  // generate A
  for( int i = 0; i < M * K; i++ ){
    h_A[i] = i / 666;
  }

  // generate B
  for( int i = 0; i < K * N; i++ ) {
    h_B[i] = i % 666;
  }

  checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy( d_B, h_B, bytes_B, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  float msecTotal = 0;
  int nIter = 1000;

  checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(start));
  for (int run = 0 ; run < nIter; run ++ ) {
    dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
    dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
    Sgemm<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X>
    <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
  }
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


  checkCudaErrors(hipMemcpy( h_C, d_C, bytes_C, hipMemcpyDeviceToHost));

  msecPerMatrixMul[0] = msecTotal / nIter;
  gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
  printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
          gigaFlops[0],
          msecPerMatrixMul[0],
          flopsPerMatrixMul);

  // cublas
  hipblasHandle_t blas_handle;
  hipblasCreate(&blas_handle);
  float alpha = 1.0;
  float beta = 0;
  checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));

  // warmup
  hipblasSgemm (blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
               M, N, K, &alpha,
               d_A, K, d_B, N, &beta, d_C, N
  );
  checkCudaErrors(hipEventRecord(start));
  for (int run = 0 ; run < nIter; run ++ ) {
    hipblasSgemm (blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                 M, N, K, &alpha,
                 d_A, K, d_B, N, &beta, d_C, N
    );
  }
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  checkCudaErrors(hipMemcpy( h_C1, d_C, bytes_C, hipMemcpyDeviceToHost));

  msecPerMatrixMul[1] = msecTotal / nIter;
  gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
  printf( "CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
          gigaFlops[1],
          msecPerMatrixMul[1],
          flopsPerMatrixMul);

  hipblasDestroy(blas_handle);

  double eps = 1.e-6;  // machine zero
  bool correct = true;
  for (int i = 0; i < M * N; i++) {
    int row = i / N;
    int col = i % N;
    double abs_err = fabs(h_C[i] - h_C1[col * M + row]);
    double dot_length = M;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;
    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
             i, h_C[i], h_C1[col * M + row], eps);
      correct = false;
      break;
    }
  }

  printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
  printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);

  // Free Memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C1);
}
